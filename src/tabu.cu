#include "hip/hip_runtime.h"
#include "tabu.hpp"

// __global__ void FillBest(int altura, int largura, individual* d_sBest, double porosity, double permeability_1, double permeability_2, double permeability_3){
//     int i = threadIdx.x + blockIdx.x * blockDim.x; 
//     int j = threadIdx.y + blockIdx.y * blockDim.y;

//     if((i < altura) && (j < largura)){
//         *d_sBest.porosity[i*largura+j] = porosity;
//         *d_sBest.permeability[i*largura+j].permeability_1 = permeability_1;
//         *d_sBest.permeability[i*largura+j].permeability_2 = permeability_2;
//         *d_sBest.permeability[i*largura+j].permeability_3 = permeability_3;
//     }
// }

void tabu_search::FirstSimulation(){
    srand((unsigned)time(0));

    CreateResultDir(0);

    double porosity = Rand_double(MIN_POROSITY, MAX_POROSITY);
    double permeability_1 = Rand_double(MIN_PERMEABILITY, MAX_PERMEABILITY);
    double permeability_2 = Rand_double(MIN_PERMEABILITY, MAX_PERMEABILITY);
    double permeability_3 = Rand_double(MIN_PERMEABILITY, MAX_PERMEABILITY);

    for(int i = 0; i < HEIGHT; i++){
        for(int j = 0; j < WIDTH; j++){
            this->sBest.porosity[i*WIDTH+j] = porosity;
            this->sBest.permeability[i*WIDTH+j].permeability_1 = permeability_1;
            this->sBest.permeability[i*WIDTH+j].permeability_2 = permeability_2;
            this->sBest.permeability[i*WIDTH+j].permeability_3 = permeability_3;
        }
    }

    WriteSimulationFile(0, 0, simulationFile, fileName, this->sBest);
    
    // Simulation(0, 1, fileName);
    // sBest.error_rank = Fitness(0, 0, this->sBest);

    // WriteErrorFile(0, this->sBest);

    // this->tabuList.push_back(sBest);

    // this->bestCandidate = sBest;

    // this->bestCandidate.proximity = ProximityFunction(this->bestCandidate);

    // Superiorization(this->bestCandidate);
}

void tabu_search::Init(){
    CreateOutputDir();

    string oilInputResult = ReadFileInput(inputOil);
    string waterInputResult = ReadFileInput(inputWater);
    string gasInputResult = ReadFileInput(inputGas);

    this->realResults = ConvertStringInputToDoubleResult(waterInputResult, oilInputResult, gasInputResult); 

    for(int i = 0; i < HEIGHT; i++){
        for(int j = 0; j < WIDTH; j++){
            this->suavityImage[i*WIDTH+j] = 5;
        }
    }

    FirstSimulation();
    // int count = 1;
    // while(count <= N_ITERATIONS && this->sBest.error_rank > STOP){
    //     OthersSimulations(count);
    //     count++;
    // }

    // SaveTabuList();

    // SaveBest();

}