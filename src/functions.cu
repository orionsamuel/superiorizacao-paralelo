#include "hip/hip_runtime.h"
#include "../include/functions.hpp"

__global__ void FitnessWater(result *d_results, result *d_simulateResults, int N, double* d_rank){
    int i = threadIdx.x;

    if(i < N){
        *d_rank += pow((d_results[i].water - d_simulateResults[i].water),2);
    }
}

__global__ void FitnessOil(result *d_results, result *d_simulateResults, int N, double* d_rank){
    int i = threadIdx.x;

    if(i < N){
        *d_rank += pow((d_results[i].oil - d_simulateResults[i].oil),2);
        printf(" Rank: %f. Óleo Real: %f. Óleo Simulado: %f.\n", d_rank, d_results[i].oil, d_simulateResults[i].oil);
    }
    
}

__global__ void FitnessGas(result *d_results, result *d_simulateResults, int N, double* d_rank){
    int i = threadIdx.x;

    if(i < N){
        *d_rank += pow((d_results[i].gas - d_simulateResults[i].gas),2);
    }
}

void functions::Simulation(int idIteration, int size, string file){
    system(Command("cp ../summaryplot.py ../Output/"+to_string(idIteration)));

    for(int i = 0; i < size; i++){
        cout << "Executando a simulação no indivíduo " << i << " da iteração " << idIteration << endl;
        system(Command("mpirun -np 4 flow ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".DATA >> out.txt"));
        system(Command("python3 ../Output/"+to_string(idIteration)+"/summaryplot.py WOPR:PROD WWPR:PROD WGPR:PROD ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".DATA >> out.txt"));
        system(Command("mv WOPR:PROD.txt ../Output/"+to_string(idIteration)+"/oleo/"+to_string(i)+".txt"));
        system(Command("mv WWPR:PROD.txt ../Output/"+to_string(idIteration)+"/agua/"+to_string(i)+".txt"));
        system(Command("mv WGPR:PROD.txt ../Output/"+to_string(idIteration)+"/gas/"+to_string(i)+".txt"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".DBG"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".EGRID"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".INFOSTEP"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".INIT"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".PRT"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".SMSPEC"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".UNRST"));
        system(Command("rm ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+file+".UNSMRY"));
    }

    system(Command("rm  ../Output/"+to_string(idIteration)+"/summaryplot.py"));
}

double functions::Rand_double(double min, double max){
    random_device rd;
    default_random_engine eng(rd());
    uniform_real_distribution<double>distr(min, max);

    double num = distr(eng);
    
    num = floor(num *100) / 100;

    return num;
}

double functions::Max(double num1, double num2){
    if(num1 > num2){
        return num1;
    }else{
        return num2;
    }
}

double functions::Min(double num1, double num2){
    if(num1 < num2){
        return num1;
    }else{
        return num2;
    }
}

const vector<string> functions::split(const string& s, const char& c){
    string buff{""};
	vector<string> v;
	
	for(auto n:s){
		if(n != c) buff+=n; else
		if(n == c && buff != "") { v.push_back(buff); buff = ""; }
	}

	if(buff != "") v.push_back(buff);
	
	return v;
}

void functions::CreateOutputDir(){
    DIR* dp = opendir("../Output/");

    if(dp == NULL){
        system("mkdir ../Output/");
    }else{
        system("rm -r -f ../Output/*");
    }
}

string functions::ReadFileInput(string file){
    ifstream fileStream(file, ios::in);
    
    string line, content;

    while(!fileStream.eof()){
        getline(fileStream, line);
        content += line;
        content += " ";
    }

    fileStream.close();

    return content;
}

vector<result> functions::ConvertStringInputToDoubleResult(string water, string oil, string gas){
    vector<string> waterSplit{split(water, ' ')};
    vector<string> oilSplit{split(oil, ' ')};
    vector<string> gasSplit{split(gas, ' ')};

    vector<result> results;
    
    for(int i = 0; i < waterSplit.size(); i++){
        result partialResult;
        partialResult.water = stod(waterSplit[i]);
        partialResult.oil = stod(oilSplit[i]);
        partialResult.gas = stod(gasSplit[i]);
        results.push_back(partialResult);
    }

    return results;

}

const char* functions::Command(string inputCommand){
    const char* command = (char*) inputCommand.c_str();
    return command;
}

void functions::CreateResultDir(int idIteration){
    string command = "../Output/"+to_string(idIteration);
    const char* file = (char*) command.c_str();
    DIR* dp = opendir(file);

    if(dp == NULL){;
        system(Command("mkdir ../Output/"+to_string(idIteration)));
        system(Command("mkdir ../Output/"+to_string(idIteration)+"/oleo"));
        system(Command("mkdir ../Output/"+to_string(idIteration)+"/agua"));
        system(Command("mkdir ../Output/"+to_string(idIteration)+"/gas"));
    }else{
        const char* rm = Command("rm -f ../Output/"+to_string(idIteration)+"/*");
        system(rm);
    }

}

void functions::WriteSimulationFile(int idIteration, int iterator, string inputFile, string file, individual sCandidate){
    ifstream input(inputFile, ios::in);
    ofstream output("../Output/"+to_string(idIteration)+"/"+to_string(iterator)+"-"+file+".DATA", ios::out);
    string line;
    int count = 0;

    while(!input.eof()){
        getline(input, line);
        if(count == 92){
            output << "    " << TOTAL_CELLS << "*" << sCandidate.porosity[0] << " /"  << endl;
        }else if(count == 96){
            output << "    " << "100*" << sCandidate.permeability[0].permeability_1 << " 100*" << sCandidate.permeability[0].permeability_2 << " 100*" << sCandidate.permeability[0].permeability_3 << " /" << endl;
        }else if(count == 100){
            output << "    " << "100*" << sCandidate.permeability[0].permeability_1 << " 100*" << sCandidate.permeability[0].permeability_2 << " 100*" << sCandidate.permeability[0].permeability_3 << " /"  << endl;
        }else if(count == 105){
            output << "    " << "100*" << sCandidate.permeability[0].permeability_1 << " 100*" << sCandidate.permeability[0].permeability_2 << " 100*" << sCandidate.permeability[0].permeability_3 << " /"  << endl;
        }else{
            output << line << endl;
        }

        count++;
    }

    input.close();
    output.close();
    
}

void functions::WriteErrorFile(int idIteration, individual sCandidate){
    ofstream errorFile("../Output/"+to_string(idIteration)+"/error.txt", ios::app);
    
    errorFile << sCandidate.error_rank << endl;

    errorFile.close();
}

double functions::activationFunction(string waterOutputResult, string oilOutputResult, string gasOutputResult, vector<result> results, int idIteration){
    double rank, rank_temp;

    string waterResult = ReadFileInput(waterOutputResult);
    string oilResult = ReadFileInput(oilOutputResult);
    string gasResult = ReadFileInput(gasOutputResult);

    vector<result> simulateResults;

    simulateResults = ConvertStringInputToDoubleResult(waterResult, oilResult, gasResult);

    int size = simulateResults.size();

    result *d_results;
    result *d_simulateResults;
    double *d_rank;

    hipMalloc((void **)&d_results, size * sizeof(result));
    hipMalloc((void **)&d_simulateResults, size * sizeof(result));
    hipMalloc((void **)&d_rank, sizeof(double));

    hipMemcpy(d_results, results.data(), size * sizeof(result), hipMemcpyHostToDevice);
    hipMemcpy(d_simulateResults, simulateResults.data(), size * sizeof(result), hipMemcpyHostToDevice);

    FitnessWater<<<1,size>>>(d_results, d_simulateResults, size, d_rank);
    hipDeviceSynchronize();

    hipMemcpy(&rank_temp, d_rank, sizeof(double), hipMemcpyDeviceToHost); 

    rank += rank_temp;
    rank *= WATER_WEIGHT;

    hipMemcpy(d_results, results.data(), size * sizeof(result), hipMemcpyHostToDevice);
    hipMemcpy(d_simulateResults, simulateResults.data(), size * sizeof(result), hipMemcpyHostToDevice);

    FitnessOil<<<1,size>>>(d_results, d_simulateResults, size, d_rank);
    hipDeviceSynchronize();

    hipMemcpy(&rank_temp, d_rank, sizeof(double), hipMemcpyDeviceToHost);

    cout << "Rank Oil: " << rank_temp << endl;

    rank += rank_temp;
    rank *= OIL_WEIGHT;

    hipMemcpy(d_results, results.data(), size * sizeof(result), hipMemcpyHostToDevice);
    hipMemcpy(d_simulateResults, simulateResults.data(), size * sizeof(result), hipMemcpyHostToDevice);

    FitnessGas<<<1,size>>>(d_results, d_simulateResults, size, d_rank);
    hipDeviceSynchronize();

    hipMemcpy(&rank_temp, d_rank, sizeof(double), hipMemcpyDeviceToHost);

    rank += rank_temp;
    rank *= GAS_WEIGHT;

    hipFree(d_results);
    hipFree(d_simulateResults);
    hipFree(d_rank);

    rank = sqrt((rank / (simulateResults.size() * N_METRICS)));

    return rank;
}
